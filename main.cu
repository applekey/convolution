#include "hip/hip_runtime.h"
#include "waveletFilter.h"
#include "helper.h"
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <chrono>
#include <cassert>
#include <unistd.h>
#include "test2D.h"

/*--------------------------------------*/
// All the code below is to test 1D signal,
// the code to test 2D is in test2D.h
/*--------------------------------------*/
using namespace std;

int64 SIGNAL_LENGTH = 0;
int64 COMPRESSION_LEVELS = 0;
int64 PRINT_INTERMEDIATE = 0;

//signal
double * host_signal_array = 0;
double * device_signal_array = 0;

//output
double * host_output_array = 0;
double * device_output_array = 0;

//low filters
double * host_low_filter_array = 0;
double * device_low_filter_array = 0;

//high filters
double * host_high_filter_array = 0;
double * device_high_filter_array = 0;

//low reconstruct filters
double * host_low_reconstruct_filter_array = 0;
double * device_low_reconstruct_filter_array = 0;

//high reconstruct filters
double * host_high_reconstruct_filter_array = 0;
double * device_high_reconstruct_filter_array = 0;

//reconstructed signal
double * host_reconstruct_output_array = 0;
double * device_reconstruted_output_array = 0;

waveletFilter filter;

void initSignal() {

    int64 num_bytes = SIGNAL_LENGTH * sizeof(double);
    assert(num_bytes != 0);

    host_signal_array = (double *)malloc(num_bytes);

    for (int64 i = 0; i < SIGNAL_LENGTH; i++) {
        /*host_signal_array[i] = 1.0 * sin((double)i /100.0) * 100.0;*/
        /*host_signal_array[i] = 0.1 * float(i);*/
        host_signal_array[i] = 1.0;
    }
}

void copyInputSignal() {

    int64 num_bytes = SIGNAL_LENGTH * sizeof(double);
    hipError_t err = hipMalloc((void **)&device_signal_array, num_bytes);

    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipMemcpy(device_signal_array, host_signal_array, num_bytes, hipMemcpyHostToDevice);
}

void initReconstructedSignal() {
    int64 num_bytes = SIGNAL_LENGTH * sizeof(double);
    hipError_t err = hipMalloc((void **)&device_reconstruted_output_array, num_bytes);

    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
}

void initOutput(int64 outputLength) {
    int64 num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);
    hipError_t err = hipMalloc((void **)&device_output_array, num_bytes);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
}

void initLowFilter() {
    int64 lowFilterLenght = 9;
    int64 num_bytes = lowFilterLenght * sizeof(double);

    host_low_filter_array = (double *)malloc(num_bytes);

    filter.getLowPassFilter(host_low_filter_array);

    hipMalloc((void **)&device_low_filter_array, num_bytes);

    hipMemcpy(device_low_filter_array, host_low_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initHighFilter() {
    int64 highFilterLenght = 9;
    int64 num_bytes = highFilterLenght * sizeof(double);

    host_high_filter_array = (double *)malloc(num_bytes);

    filter.getHighPassFilter(host_high_filter_array);
    hipMalloc((void **)&device_high_filter_array, num_bytes);

    hipMemcpy(device_high_filter_array, host_high_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initLowReconstructFilter() {
    int64 lowFilterLenght = 9;
    int64 num_bytes = lowFilterLenght * sizeof(double);

    host_low_reconstruct_filter_array = (double *)malloc(num_bytes);

    filter.getLowReconstructFilter(host_low_reconstruct_filter_array);
    hipMalloc((void **)&device_low_reconstruct_filter_array, num_bytes);

    hipMemcpy(device_low_reconstruct_filter_array, host_low_reconstruct_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initHighReconstructFilter() {
    int64 highFilterLenght = 9;
    int64 num_bytes = highFilterLenght * sizeof(double);

    host_high_reconstruct_filter_array = (double *)malloc(num_bytes);

    filter.getHighReconstructFilter(host_high_reconstruct_filter_array);
    hipMalloc((void **)&device_high_reconstruct_filter_array, num_bytes);

    hipMemcpy(device_high_reconstruct_filter_array, host_high_reconstruct_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void transferMemoryBack(int64 outputLength) {
    outputLength -= SIGNAL_LENGTH / 2;
    int64 num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);

    /*hipHostAlloc((void**)&host_output_array, num_bytes, */
    /*hipHostMallocDefault) ;*/

    host_output_array = (double *)malloc(num_bytes);
    hipMemcpy(host_output_array, device_output_array + SIGNAL_LENGTH / 2, num_bytes, hipMemcpyDeviceToHost);
}

void transferReconstructedMemoryBack(int64 outputLength) {
    int64 num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);

    host_reconstruct_output_array = (double *)malloc(num_bytes);
    hipMemcpy(host_reconstruct_output_array, device_reconstruted_output_array,
               num_bytes, hipMemcpyDeviceToHost);
}

void printOutputCoefficients(double * hostOutput, MyVector & coefficientIndicies) {
    int64 offset = 0;
    /*int offset = SIGNAL_LENGTH / 2;*/
    int coefficientLevels = coefficientIndicies.size();

    /*int total = coefficientIndicies[3];*/
    /*std::cerr<<coefficientLevels<<" "<<total<<std::endl;*/
    /*for(int i =0; i< total; i++) {*/
    /*std::cerr<<hostOutput[offset + i]<<std::endl;*/
    /*}*/

    for (int i = 0; i < coefficientLevels - 1; i++) {
        std::cerr << "Level: " << i << std::endl;
        int64 levelCoefficientIndex = coefficientIndicies[i];
        int64 numberOfCoefficents = coefficientIndicies[i + 1] - coefficientIndicies[i];

        for (int64 j = 0; j < numberOfCoefficents; j++) {
            double coeffVal = hostOutput[levelCoefficientIndex + j + offset];
            std::cerr << coeffVal << " ";
        }
        std::cerr << std::endl;
    }
}

void printReconstructedSignal() {
    std::cerr << "Reconstructed Signal" << std::endl;
    for (int64 i = 0 ; i < SIGNAL_LENGTH; i++) {
        std::cerr << host_reconstruct_output_array[i] << " ";
    }
    std::cerr << std::endl;
}
bool isCloseTo(double a, double b, double epsilon) {
    if (abs(a - b) < epsilon) {
        return true;
    } else {
        return false;
    }
}
void verifyReconstructedSignal() {
    bool allCorrect = true;
    std::cerr << "Verifiying Signal" << std::endl;
    for (int64 i = 0 ; i < SIGNAL_LENGTH; i++) {
        if (!isCloseTo(host_reconstruct_output_array[i], 1, 0.01)) {
            allCorrect = false;
        }
    }

    if(allCorrect) {
        std::cerr<<"all correct 1D"<<std::endl;
    } else {
        std::cerr<<"reconstruction error 1D"<<std::endl;
    }
}

void freeMemory() {
    free(host_signal_array);
    free(host_output_array);

    hipFree(device_signal_array);
    hipFree(device_output_array);

    free(host_low_filter_array);
    hipFree(device_low_filter_array);

    free(host_high_filter_array);
    hipFree(device_high_filter_array);

    free(host_low_reconstruct_filter_array);
    hipFree(device_low_reconstruct_filter_array);

    free(host_high_reconstruct_filter_array);
    hipFree(device_high_reconstruct_filter_array);
}

void writeResultsToMemory(double * output, int64 length) {
    /*double epsilon = 0.0000001;*/
    /*double a = -1.41442e-12;*/
    /*double b = 1.41421;*/

    /*int offset = SIGNAL_LENGTH / 2;*/
    /*for(int i = 0; i < length/2; i++) {*/
    /*if(abs(a -  output[i + offset]) < epsilon * 1.0e-12 ) {*/
    /*std::cerr<<"error "<<output[i + offset]<<std::endl;*/
    /*}*/
    /*}*/
    /*for(int i = length/2; i < length; i++) {*/
    /*if(abs(b -  output[i + offset]) < epsilon) {*/
    /*std::cerr<<"error "<<output[i + offset]<<std::endl;*/
    /*}*/
    /*}*/
    /*return;*/
    int64 offset = SIGNAL_LENGTH / 2;
    ofstream myfile;
    myfile.open("output.txt");

    for (int64 i = 0; i < length; i++) {
        myfile << output[i + offset] << "\n";
    }
    myfile.close();
}

double * initTmpCoefficientMemory(int64 signalLength) {
    double * lowCoefficientMemory = 0;
    int64 num_bytes = signalLength * sizeof(double);
    assert(num_bytes != 0);
    hipMalloc((void **)&lowCoefficientMemory, num_bytes);
    return lowCoefficientMemory;
}

void test1D() {
    std::cerr << "Testing 1D Decompose" << std::endl;
    MyVector coefficientIndicies;

    int64 outputLength = calculateCoefficientLength(coefficientIndicies, COMPRESSION_LEVELS, SIGNAL_LENGTH);
    outputLength += SIGNAL_LENGTH / 2; //add extra for buffer for first low coefficient

    filter.constructFilters();
    initLowFilter();
    initHighFilter();
    initLowReconstructFilter();
    initHighReconstructFilter();
    initSignal();
    initOutput(outputLength);
    initReconstructedSignal();

    int64 extendedSignalLength = SIGNAL_LENGTH + (SIGNAL_LENGTH / 2 ) * 2;
    double * tmpMemoryDWT = initTmpCoefficientMemory(extendedSignalLength);

    copyInputSignal();
    auto startDecompose = std::chrono::system_clock::now();
    /*-------------------COMPRESS THE SIGNAL---------------------*/
    //run filter
    dwt(coefficientIndicies, COMPRESSION_LEVELS,
        device_signal_array, SIGNAL_LENGTH,
        device_low_filter_array, device_high_filter_array,
        device_output_array, tmpMemoryDWT, 9);

    //transfer output back

    hipDeviceSynchronize();
    auto endDecompose = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = endDecompose - startDecompose;
    std::cout << diff.count() << "  1D Compression Total s\n";
    transferMemoryBack(outputLength);

    hipFree(tmpMemoryDWT);

    if(PRINT_INTERMEDIATE) {
        printOutputCoefficients(host_output_array, coefficientIndicies);
    };

    /*-------------------UN-COMPRESS THE SIGNAL---------------------*/
    double * tmpMemoryDWTHigh = initTmpCoefficientMemory(SIGNAL_LENGTH);
    double * tmpMemoryDWTLow = initTmpCoefficientMemory(SIGNAL_LENGTH);

    auto startReconstruct = std::chrono::system_clock::now();
    iDwt(coefficientIndicies, COMPRESSION_LEVELS,
         SIGNAL_LENGTH, 9, device_output_array + SIGNAL_LENGTH / 2,
         device_low_reconstruct_filter_array,
         device_high_reconstruct_filter_array,
         device_reconstruted_output_array,
         tmpMemoryDWTHigh, tmpMemoryDWTLow);
    hipDeviceSynchronize();
    auto endReconstruct = std::chrono::system_clock::now();

    diff = endReconstruct - startReconstruct;
    std::cout << diff.count() << "  1D De-Compression Total s\n";
    transferReconstructedMemoryBack(SIGNAL_LENGTH);
    verifyReconstructedSignal();

    if(PRINT_INTERMEDIATE) {
        printReconstructedSignal();
    };

    /*-------------------CLEAN-UP---------------------*/
    //done free memory
    hipFree(tmpMemoryDWTHigh);
    hipFree(tmpMemoryDWTLow);
    freeMemory();
}

void verifyTimer() {
    std::cerr<<"Testing timer, should be 1 second"<<std::endl;
    auto start = std::chrono::system_clock::now();
    usleep(1000000);
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << "timer reported: "<<diff.count() << " s\n";
}

int isPowerOfTwo (unsigned int x)
{
 while (((x % 2) == 0) && x > 1) /* While x is even and > 1 */
   x /= 2;
 return (x == 1);
}

int main(int argc, const char * argv[]) {
    //scrub input
    verifyTimer();

    if(argc <= 4) {
        std::cerr<<"incorrect args, example is ./wave 16384 3 1, args are signal size, level of compression, test number"<<std::endl;
        return 0;
    }

    int N = atoi(argv[1]);
    int levels = atoi(argv[2]);
    int test = atoi(argv[3]);
    int printIntermediate = atoi(argv[4]);
    assert(N > 0);
    assert(levels > 0);
    assert(test > 0);
    assert(printIntermediate == 0 ||  printIntermediate == 1);

#if defined SHARED_MEMORY
    std::cerr<<"Running with shared memory optimization"<<std::endl;
#endif
#if defined BIG 
    std::cerr<<"Running large sizes >= 1024"<<std::endl;
#endif

    if(!isPowerOfTwo(N)) {
        std::cerr<<"N,"<<N<<" is not a power of 2"<<std::endl;
        return 0;
    }

    std::cerr<<"N is: "<<N<<" levels is: "<<levels<<std::endl;
    SIGNAL_LENGTH = N;
    COMPRESSION_LEVELS = levels;
    PRINT_INTERMEDIATE = printIntermediate;
    
    if(test == 1) {
        test1D();
    } else if(test == 2) {
        test2D(SIGNAL_LENGTH, COMPRESSION_LEVELS);
    }
    return 0;
}
