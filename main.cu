#include "hip/hip_runtime.h"
#include "waveletFilter.h"
#include "helper.h"
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <chrono>
#include <cassert>

/*#define SIGNAL_LENGTH 134217728 */
/*#define SIGNAL_LENGTH 67108864 */
/*#define SIGNAL_LENGTH 1048576 */
#define SIGNAL_LENGTH 524288 
/*#define SIGNAL_LENGTH  32 */
#define COMPRESSION_LEVELS 3

using namespace std;

//signal
double * host_signal_array = 0;
double * device_signal_array = 0;

//output
double * host_output_array = 0;
double * device_output_array = 0;

//low filters
double * host_low_filter_array = 0;
double * device_low_filter_array = 0;

//high filters
double * host_high_filter_array = 0;
double * device_high_filter_array = 0;

//low reconstruct filters
double * host_low_reconstruct_filter_array = 0;
double * device_low_reconstruct_filter_array = 0;

//high reconstruct filters
double * host_high_reconstruct_filter_array = 0;
double * device_high_reconstruct_filter_array = 0;

//reconstructed signal
double * host_reconstruct_output_array = 0;
double * device_reconstruted_output_array = 0;

waveletFilter filter;

std::vector<int> coefficientIndicies; 

void initSignal() {

    long long num_bytes = SIGNAL_LENGTH * sizeof(double);
    assert(num_bytes != 0);

    host_signal_array = (double*)malloc(num_bytes);

    for(int i = 0; i < SIGNAL_LENGTH; i++) {
        /*host_signal_array[i] = 1.0 * sin((double)i /100.0) * 100.0;*/
        host_signal_array[i] = 1.0;
    }
}

void copyInputSignal() {

    long long num_bytes = SIGNAL_LENGTH * sizeof(double);
    hipError_t err = hipMalloc((void**)&device_signal_array, num_bytes);

    if(err != hipSuccess){
         printf("The error is %s", hipGetErrorString(err));
    }
    hipMemcpy(device_signal_array, host_signal_array, num_bytes, hipMemcpyHostToDevice);
}

void initReconstructedSignal() {
    long long num_bytes = SIGNAL_LENGTH * sizeof(double);
    hipError_t err = hipMalloc((void**)&device_reconstruted_output_array, num_bytes);

    if(err != hipSuccess){
         printf("The error is %s", hipGetErrorString(err));
    }
}

void initOutput(int outputLength) {
    long long num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);
    hipError_t err = hipMalloc((void**)&device_output_array, num_bytes);
    if(err != hipSuccess){
         printf("The error is %s", hipGetErrorString(err));
    }
}

void initLowFilter() {
    int lowFilterLenght = 9;
    long long num_bytes = lowFilterLenght * sizeof(double);

    host_low_filter_array = (double*)malloc(num_bytes);

    filter.getLowPassFilter(host_low_filter_array);

    hipMalloc((void**)&device_low_filter_array, num_bytes);

    hipMemcpy(device_low_filter_array, host_low_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initHighFilter() {
    int highFilterLenght = 9;
    long long num_bytes = highFilterLenght * sizeof(double);

    host_high_filter_array = (double*)malloc(num_bytes);

    filter.getHighPassFilter(host_high_filter_array);
    hipMalloc((void**)&device_high_filter_array, num_bytes);

    hipMemcpy(device_high_filter_array, host_high_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initLowReconstructFilter() {
    int lowFilterLenght = 9;
    long long num_bytes = lowFilterLenght * sizeof(double);

    host_low_reconstruct_filter_array = (double*)malloc(num_bytes);

    filter.getLowReconstructFilter(host_low_reconstruct_filter_array);
    hipMalloc((void**)&device_low_reconstruct_filter_array, num_bytes);

    hipMemcpy(device_low_reconstruct_filter_array, host_low_reconstruct_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void initHighReconstructFilter() {
    int highFilterLenght = 9;
    long long num_bytes = highFilterLenght * sizeof(double);

    host_high_reconstruct_filter_array = (double*)malloc(num_bytes);

    filter.getHighReconstructFilter(host_high_reconstruct_filter_array);
    hipMalloc((void**)&device_high_reconstruct_filter_array, num_bytes);

    hipMemcpy(device_high_reconstruct_filter_array, host_high_reconstruct_filter_array, num_bytes, hipMemcpyHostToDevice);
}

void transferMemoryBack(int outputLength) {
    outputLength -=SIGNAL_LENGTH / 2; 
    long long num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);

    host_output_array = (double*)malloc(num_bytes);
    hipMemcpy(host_output_array, device_output_array + SIGNAL_LENGTH / 2, num_bytes, hipMemcpyDeviceToHost);  
    /*hipMemcpy(host_output_array, device_output_array, num_bytes, hipMemcpyDeviceToHost);  */
}

void transferReconstructedMemoryBack(int outputLength) {
    long long num_bytes = outputLength * sizeof(double);
    assert(num_bytes != 0);

    host_reconstruct_output_array = (double*)malloc(num_bytes);
    hipMemcpy(host_reconstruct_output_array, device_reconstruted_output_array, 
               num_bytes, hipMemcpyDeviceToHost);  
}

void printOutputCoefficients(double * hostOutput, std::vector<int> coefficientIndicies) {
    int offset = 0;
    /*int offset = SIGNAL_LENGTH / 2;*/
    int coefficientLevels = coefficientIndicies.size();

    /*int total = coefficientIndicies[3];*/
    /*std::cerr<<coefficientLevels<<" "<<total<<std::endl;*/
    /*for(int i =0; i< total; i++) {*/
        /*std::cerr<<hostOutput[offset + i]<<std::endl;*/
    /*}*/
    
    for(int i = 0; i < coefficientLevels - 1;i++) {
        std::cerr<<"Level: "<<i<<std::endl;
        int levelCoefficientIndex = coefficientIndicies[i];
        int numberOfCoefficents = coefficientIndicies[i + 1] - coefficientIndicies[i];

        for(int j = 0; j < numberOfCoefficents; j++) {
            double coeffVal = hostOutput[levelCoefficientIndex + j + offset];
            std::cerr<<coeffVal<<" ";
        }
        std::cerr<<std::endl;
    }
}

void printReconstructedSignal() {
    std::cerr<<"Reconstructed Signal"<<std::endl;
    for(int i = 0 ; i< SIGNAL_LENGTH; i++) {
        std::cerr<<host_reconstruct_output_array[i]<<" ";
    } 
    std::cerr<<std::endl;
}
bool isCloseTo(double a, double b, double epsilon) {
    if(abs(a-b) < epsilon) {
        return true;
    } else {
        return false;
    }
}
void verifyReconstructedSignal() {
    bool allCorrect = true;
    std::cerr<<"Verifiying Signal"<<std::endl;
    for(int i = 0 ; i< SIGNAL_LENGTH; i++) {
        if(!isCloseTo(host_reconstruct_output_array[i],1, 0.0001)) {
         std::cerr<<host_reconstruct_output_array[i]<<std::endl;
          allCorrect = false;  
        }
    } 
    assert(allCorrect);
}

void freeMemory() {
    free(host_signal_array);
    free(host_output_array);

    hipFree(device_signal_array);
    hipFree(device_output_array);

    free(host_low_filter_array);
    hipFree(device_low_filter_array);

    free(host_high_filter_array);
    hipFree(device_high_filter_array);

    free(host_low_reconstruct_filter_array);
    hipFree(device_low_reconstruct_filter_array);

    free(host_high_reconstruct_filter_array);
    hipFree(device_high_reconstruct_filter_array);
}

void writeResultsToMemory(double * output, int length) {
    /*double epsilon = 0.0000001;*/
    /*double a = -1.41442e-12;*/
    /*double b = 1.41421;*/

    /*int offset = SIGNAL_LENGTH / 2;*/
    /*for(int i = 0; i < length/2; i++) {*/
        /*if(abs(a -  output[i + offset]) < epsilon * 1.0e-12 ) {*/
            /*std::cerr<<"error "<<output[i + offset]<<std::endl;*/
        /*}*/
    /*}*/
    /*for(int i = length/2; i < length; i++) {*/
        /*if(abs(b -  output[i + offset]) < epsilon) {*/
            /*std::cerr<<"error "<<output[i + offset]<<std::endl;*/
        /*}*/
    /*}*/
    /*return;*/
    int offset = SIGNAL_LENGTH / 2;
    ofstream myfile;
    myfile.open("output.txt");
    
    for(int i = 0; i < length; i++) {
        myfile << output[i + offset]<<"\n";
    }
    myfile.close();
}


int main(int argc, const char * argv[]) {
    int outputLength = calculateCoefficientLength(coefficientIndicies, COMPRESSION_LEVELS, SIGNAL_LENGTH);
    outputLength += SIGNAL_LENGTH / 2; //add extra for buffer for first low coefficient

    filter.constructFilters();
    initLowFilter();
    initHighFilter();
    initLowReconstructFilter();
    initHighReconstructFilter();
    initSignal();
    initOutput(outputLength);
    initReconstructedSignal();

/*-------------------COMPRESS THE SIGNAL---------------------*/
auto startDecompose = std::chrono::system_clock::now();
    copyInputSignal();
    //run filter   
    dwt(coefficientIndicies, COMPRESSION_LEVELS, 
        device_signal_array, SIGNAL_LENGTH,
        device_low_filter_array, device_high_filter_array,
        device_output_array, 9);

    //transfer output back
    transferMemoryBack(outputLength);

auto endDecompose = std::chrono::system_clock::now();
std::chrono::duration<double> diff = endDecompose-startDecompose;
std::cout<< diff.count() << " s\n";
    /*printOutputCoefficients(host_output_array, coefficientIndicies);*/

    /*int ab = calculateCoefficientLength(coefficientIndicies, COMPRESSION_LEVELS, SIGNAL_LENGTH);*/
    /*writeResultsToMemory(host_output_array, ab);*/

/*-------------------DEBUG---------------------*/
/*std::cerr<<"low"<<std::endl;*/
/*for(int i = 0;i < 9; i++ ) {*/
    /*std::cerr<<host_low_reconstruct_filter_array[i]<<std::endl;*/
/*}*/

/*std::cerr<<"break"<<std::endl;*/

/*std::cerr<<"high"<<std::endl;*/
/*for(int i = 0;i < 9; i++ ) {*/
    /*std::cerr<<host_high_reconstruct_filter_array[i]<<std::endl;*/
/*}*/

/*-------------------UN-COMPRESS THE SIGNAL---------------------*/
auto startReconstruct = std::chrono::system_clock::now();
    iDwt(coefficientIndicies, COMPRESSION_LEVELS, 
         SIGNAL_LENGTH, 9, device_output_array + SIGNAL_LENGTH / 2,
         device_low_reconstruct_filter_array,
         device_high_reconstruct_filter_array,
         device_reconstruted_output_array);

    transferReconstructedMemoryBack(SIGNAL_LENGTH);
auto endReconstruct = std::chrono::system_clock::now();
diff = endReconstruct-startReconstruct;
std::cout<< diff.count() << " s\n";
    /*printReconstructedSignal();*/
    verifyReconstructedSignal();
    /*printReconstructedSignal();*/

/*-------------------CLEAN-UP---------------------*/
    //done free memory 
    freeMemory();

    return 0;
}
